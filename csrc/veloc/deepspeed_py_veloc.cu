#include "hip/hip_runtime.h"
#include <deepspeed_py_veloc.h>

void veloc_ckpt_t::_d2h_trf() {
    checkCuda(hipSetDevice(_gpu_id));
    while (is_active) {
        try {
            std::unique_lock<std::mutex> _lock_d2h(_mutex_d2h);
            while(_pending_d2h.empty() && is_active)
                _cv_d2h.wait(_lock_d2h);
            if (!is_active) {
                _pending_d2h.clear();
                _lock_d2h.unlock();
                _cv_d2h.notify_all();
                DBG("---- Returning from d2h thread " << _gpu_id);
                return;
            }
            TIMER_START(d2h_time);
            auto e = _pending_d2h.front();
            _lock_d2h.unlock();
            _cv_d2h.notify_all();

            int version = std::get<0>(e);
            uint64_t uid = std::get<1>(e);
            std::string path = std::get<2>(e);
            torch::Tensor t = std::get<3>(e);
            size_t size = std::get<4>(e);
            size_t file_offset = std::get<5>(e);
            uint64_t enqueued_time = std::get<6>(e);
            DBG("[D2H][" << _gpu_id << "] transfer of tensor " << uid << " version " << version << " delta " << get_current_ts()-enqueued_time 
            << " enqueued at " << enqueued_time << " started at " << get_current_ts());
            mem_region_t* m = mem->allocate(uid, size);
            char *host_ptr = m->ptr;
            char *src_ptr = static_cast<char *>(t.data_ptr());
           
            
            // Previously working version with single-shot d2h transfer.
            checkCuda(hipMemcpyAsync(host_ptr, src_ptr, size, hipMemcpyDeviceToHost, _cpy_stream));
            checkCuda(hipStreamSynchronize(_cpy_stream));
            std::unique_lock<std::mutex> _lock_h2f(_mutex_h2f);
            _pending_h2f.push_back(std::make_tuple(version, m->uid, path, host_ptr, size, file_offset, get_current_ts(), true, size));
            _lock_h2f.unlock();
            _cv_h2f.notify_all();
            
            /*
            size_t rem = size;
            size_t D2H_CHUNK_SIZE = (64 << 20);
            TIMER_START(memcpy_time);
            while(rem > 0) {
                size_t chunkSize = D2H_CHUNK_SIZE < rem ? D2H_CHUNK_SIZE : rem;
                size_t curr = size - rem;
                TIMER_START(ctime);
                checkCuda(hipMemcpyAsync(host_ptr+curr, src_ptr+curr, chunkSize, hipMemcpyDeviceToHost, _cpy_stream));
                checkCuda(hipStreamSynchronize(_cpy_stream));
                TIMER_STOP(ctime, "[D2H][" << _gpu_id << "] D2H Part Memcpy time for " << m->uid << " version " << version << " rem " << rem << " out of " << size, size);
                rem -= chunkSize;
                std::unique_lock<std::mutex> _lock_h2f(_mutex_h2f);
                _pending_h2f.push_back(std::make_tuple(version, m->uid, path, host_ptr+curr, chunkSize, file_offset+curr, get_current_ts(), rem==0 //EOF?
                , size));
                _lock_h2f.unlock();
                _cv_h2f.notify_all();
            }
            TIMER_STOP(memcpy_time, "[D2H][" << _gpu_id << "] D2H Memcpy time for " << m->uid << " version " << version, size);
            */
            
            
            _lock_d2h.lock();
            _pending_d2h.pop_front();
            _lock_d2h.unlock();
            _cv_d2h.notify_all();
            TIMER_STOP(d2h_time, "[D2H][" << _gpu_id << "] Total time for GPU to process " << m->uid << " version " << version, size);
            DBG("[D2H][" << _gpu_id << "] transfer of tensor " << uid  << " version " << version << " delta " << get_current_ts()-enqueued_time << " enqueued at " << enqueued_time << " completed at " << get_current_ts());
        } catch (std::exception &e) {
            FATAL("Exception caught in d2h trf." << e.what());
        } catch (...) {
            FATAL("Unknown exception caught in d2h trf.");
        }
    }
}

void veloc_ckpt_t::_h2f_trf() {
    checkCuda(hipSetDevice(_gpu_id));
    while (is_active) {
        try {
            std::unique_lock<std::mutex> _lock_h2f(_mutex_h2f);
            TIMER_START(h2f_wait);
            while(_pending_h2f.empty() && is_active)
                _cv_h2f.wait(_lock_h2f);
            if (!is_active) {
                _lock_h2f.unlock();
                _cv_h2f.notify_all();
                DBG("---- Returning from h2f thread " << _gpu_id);
                return;
            }
            TIMER_START(h2f_time);
            auto e = _pending_h2f.front();
            _lock_h2f.unlock();
            _cv_h2f.notify_all();
            
            int version = std::get<0>(e);
            uint64_t uid = std::get<1>(e);
            std::string path = std::get<2>(e);
            char* ptr = std::get<3>(e);
            size_t size = std::get<4>(e);
            size_t file_offset = std::get<5>(e);
            uint64_t enqueued_time = std::get<6>(e);
            bool eof = std::get<7>(e);
            size_t total_size = std::get<8>(e);
            DBG("[H2F][" << _gpu_id << "] flush for tensor uid " << uid  << " version " << version << " delta " << get_current_ts()-enqueued_time << " enqueued at " << enqueued_time << " started at " << get_current_ts());


            // -------- Multi-thread working right
            // int num_writer_threads = writer_threads;
            // size_t chunkSize = ceil(size / num_writer_threads);
            // if (chunkSize < MIN_CHUNK_SIZE) {
            //     chunkSize = MIN_CHUNK_SIZE;
            //     num_writer_threads = ceil(size/chunkSize);
            // }
            // std::vector<std::thread> write_threads(num_writer_threads);
            // for(int threadID=0; threadID<num_writer_threads; threadID++) {
            //     size_t startIdx = threadID * chunkSize;
            //     size_t endIdx = (threadID == num_writer_threads - 1) ? size : startIdx + chunkSize;
            //     write_threads[threadID] = std::thread([&] { _write_file(ptr, path, startIdx, endIdx, file_offset, uid, version, threadID); });
            // }
            // for(int threadID=0; threadID<num_writer_threads; threadID++) { 
            //     write_threads[threadID].join();
            // }
            // -------- Multi-thread working right
            
            // -------- Single thread working right
            
            // -------- Multi-thread working right
            // int num_writer_threads = writer_threads;
            // size_t chunkSize = ceil(size / num_writer_threads);
            // if (chunkSize < MIN_CHUNK_SIZE) {
            //     chunkSize = MIN_CHUNK_SIZE;
            //     num_writer_threads = ceil(size/chunkSize);
            // }
            // std::vector<std::thread> write_threads(num_writer_threads);
            // for(int threadID=0; threadID<num_writer_threads; threadID++) {
            //     size_t startIdx = threadID * chunkSize;
            //     size_t endIdx = (threadID == num_writer_threads - 1) ? size : startIdx + chunkSize;
            //     write_threads[threadID] = std::thread([&] { _write_file(ptr, path, startIdx, endIdx, file_offset, uid, version, threadID); });
            // }
            // for(int threadID=0; threadID<num_writer_threads; threadID++) { 
            //     write_threads[threadID].join();
            // }
            // -------- Multi-thread working right
            
            // -------- Single thread working right
            std::ofstream f;            
            f.exceptions(std::ofstream::failbit | std::ofstream::badbit);
            f.open(path,  std::ofstream::out | std::ofstream::binary | std::ofstream::app);
            f.seekp(file_offset);
            f.write(ptr, size);
            f.close();
            
            if (eof)
                mem->deallocate(uid, total_size);
            _lock_h2f.lock();
            _pending_h2f.pop_front();
            _lock_h2f.unlock();
            _cv_h2f.notify_all();
            TIMER_STOP(h2f_time, "[H2F][" << _gpu_id << "] Total time in h2f to save tensor " << uid << " version " << version << " of size " << size, size);
            DBG("[H2F][" << _gpu_id << "] flush for tensor uid " << uid  << " version " << version << " delta " << get_current_ts()-enqueued_time << " enqueued at " << enqueued_time << " completed at " << get_current_ts());
        }  catch (std::exception &e) {
            FATAL("Exception caught in h2f trf." << e.what());
        } catch (...) {
            FATAL("Unknown exception caught in h2f trf.");
        }
    }
}

void veloc_ckpt_t::ckpt_obj(int version, const std::uint64_t start_offset, const std::uint64_t end_offset, const std::uint64_t ptr_id, const std::uint64_t size, const int device_id, const std::uint64_t file_offset, std::string path) {
    FATAL("In ckpt obj for " << version <<  " size " << size << " device " << device_id << " path " << path)
    try {
        char* ptr = reinterpret_cast<char*>(ptr_id);
        hipPointerAttribute_t attr;
        checkCuda(hipPointerGetAttributes(&attr, (const void *)ptr));
        if (attr.type == hipMemoryTypeDevice)
            assert((attr.type == hipMemoryTypeDevice && device_id > -1 && device_id == _gpu_id) && "Device pointer problem in ckpt_obj");
        if (attr.type == hipMemoryTypeDevice && device_id > -1) {
            throw std::runtime_error("Do not know how to checkpoint device objects");
            return;
        }
        throw std::runtime_error("Do not know how to checkpoint device objects");
    } catch (std::exception &e) {
        FATAL("Exception caught in ckpt_pickle." << e.what());
    } catch (...) {
        FATAL("Unknown exception caught in ckpt." << path);
    }
}

void veloc_ckpt_t::ckpt_tensor(int version, const std::uint64_t start_offset, const std::uint64_t end_offset, const torch::Tensor &t, 
        const std::uint64_t size, const int device_id, const std::uint64_t file_offset, std::string path) {
    try {
        if (t.device().is_cuda()) 
            assert((t.device().index() == _gpu_id) && "Tensor not on the same GPU as ckpt engine");
        uint64_t uid = local_uid++;
        if (t.device().is_cuda()) {
            DBG("[" << _gpu_id << "] Enqueuing GPU tensor " << uid << " version  " << version << " size " << size);
            std::unique_lock<std::mutex> _lock_d2h(_mutex_d2h);
            _pending_d2h.push_back(std::make_tuple(version, uid, path, t, size, file_offset, get_current_ts()));
            _lock_d2h.unlock();
            _cv_d2h.notify_all();
            return;
        } 
        DBG("[" << _gpu_id << "] Enqueuing host tensor " << uid << " version  " << version << " size " << size);
        std::unique_lock<std::mutex> _lock_h2f(_mutex_h2f);
        _pending_h2f.push_back(std::make_tuple(version, uid, path, static_cast<char *>(t.data_ptr()), size, file_offset, get_current_ts(), true, size));
        _lock_h2f.unlock();
        _cv_h2f.notify_all();
        return;
    } catch (std::exception &e) {
        FATAL("Exception caught in ckpt_tensor." << e.what());
    } catch (...) {
        FATAL("Unknown exception caught in ckpt_tensor." << path);
    }
}

void veloc_ckpt_t::wait(int version) {
    try {
        TIMER_START(wait_timer);
        std::unique_lock<std::mutex> _lock_d2h(_mutex_d2h);
        while(!(_pending_d2h.empty())) {
            DBG("[" << _gpu_id << "] Waiting in d2h for " << _pending_d2h.size());
            for(auto e: _pending_d2h) {
                DBG("[" << _gpu_id << "]" << std::get<0>(e) << " UID " << std::get<1>(e) << " size " << std::get<4>(e));
            }
            _cv_d2h.wait(_lock_d2h);
        }
        _lock_d2h.unlock();
        _cv_d2h.notify_all();
        TIMER_STOP(wait_timer, "[" << _gpu_id << "] Wait D2H complete ", 1);
    }  catch (std::exception &e) {
        FATAL("Exception caught in wait D2H." << e.what());
    } catch (...) {
        FATAL("Unknown exception caught in wait D2H.");
    }
}

void veloc_ckpt_t::shutdown() {
    try {
        wait();
        std::unique_lock<std::mutex> _lock_h2f(_mutex_h2f);
        // Wait for D2H transfers
        while((!_pending_h2f.empty())) {
            DBG("[" << _gpu_id << "] Waiting in h2f for " << _pending_h2f.size());
            for(auto e: _pending_h2f) {
                DBG("[" << _gpu_id << "]" << std::get<0>(e) << " UID " << std::get<1>(e) << " size " << std::get<4>(e));
            }
            _cv_h2f.wait(_lock_h2f);
        }
        _lock_h2f.unlock();
        _cv_h2f.notify_all();
        
        is_active = false;
        mem->shutdown();
        _cv_h2f.notify_all();
        _cv_d2h.notify_all();
        _thread_d2h.join();
        _thread_h2f.join();
        return;
    } catch (std::exception &e) {
        FATAL("Exception caught in shutdown." << e.what());
    } catch (...) {
        FATAL("Unknown exception caught in shutdown.");
    }
}
